float* solveGPU(float* tab, int N, int dim, int k)
{
    // Allocate memory and prepare data
    float *d_centroid, *d_tab;
    float *h_centroid = new float[dim * k];

    hipMalloc(&d_centroid, dim * k * sizeof(float));
    hipMalloc(&d_tab, dim * N * sizeof(float));
    hipMemcpy(d_tab, h_tab, N * dim * sizeof(float));
    
    hipMemcpy(h_centroid, d_centroid, dim * k * sizeof(float), hipMemcpyDeviceToHost);

    return h_centroid;
}