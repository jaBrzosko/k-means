#include "hip/hip_runtime.h"
#include <cfloat>
#include <iostream>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>



__global__ void divNewTab2(float* centroid, float* newCentroid, int* count, int k, int dim)
{
    unsigned int kIndex   = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int dimIndex = threadIdx.y + blockIdx.y * blockDim.y;

    // Don't go over boundaries
    if(kIndex >= k || dimIndex >= dim) return;

    int cnt = count[kIndex];
    if(cnt != 0)
        centroid[kIndex + dimIndex * k] = newCentroid[kIndex + dimIndex * k] / cnt;
}

__global__ void calculateBestDistance2(float* tab, float* kTab, float* kNewTab, int* membership, int* kCount, int* changed, int N, int k, int dim)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= N)
        return;
    float bestDistance = FLT_MAX;
    int bestCentroid = -1;
    
    // Check every cluster
    for(int i = 0; i < k; i++)
    {
        // Calculate distance to this cluster
        float d = 0;
        for(int j = 0; j < dim; j++)
        {
            float point = tab[tid + j * N];
            float centroid = kTab[i + j * k];
            d += (point - centroid) * (point - centroid);
        }

        // Check if it is closer
        if(d < bestDistance)
        {
            bestDistance = d;
            bestCentroid = i;
        }
    }

    changed[tid] = 0;
    // Save better cluster
    if(membership[tid] != bestCentroid)
    {
        membership[tid] = bestCentroid;
        changed[tid] = 1;
    }
    atomicAdd(&kCount[bestCentroid], 1);
    for(int i = 0; i < dim; i++)
    {
        atomicAdd(&kNewTab[bestCentroid + i * k], tab[tid + i * N]);
    }
}

float* solveGPU2(float* h_tab, int N, int dim, int k)
{
    // Allocate memory and prepare data
    float *d_centroid, *d_tab, *d_newCentroid;
    int *d_membership, *d_changed;
    int *d_count;
    float *h_centroid = new float[dim * k];

    hipMalloc(&d_centroid, dim * k * sizeof(float));
    hipMalloc(&d_newCentroid, dim * k * sizeof(float));
    hipMalloc(&d_tab, dim * N * sizeof(float));
    hipMalloc(&d_membership, N * sizeof(int));
    hipMalloc(&d_changed, N * sizeof(int));
    hipMalloc(&d_count, k * sizeof(int));



    hipMemcpy(d_tab, h_tab, N * dim * sizeof(float), hipMemcpyHostToDevice);

    // Initialize centroid positions as first k point in tab
    for(int i = 0; i < k; i++)
    {
        for(int j = 0; j < dim; j++)
        {
            h_centroid[i + k * j] = h_tab[i + N * j];
        }
    }
    hipMemcpy(d_centroid, h_centroid, k * dim * sizeof(float), hipMemcpyHostToDevice);
    
    // Solve k-means
    int block = 1024;
    int gridN = N / 1024 + (N % 1024 == 0 ? 0 : 1);
    
    dim3 gridK(k / 32 + (k % 1024 == 0 ? 0 : 1), dim / 32 + (dim % 1024 == 0 ? 0 : 1), 1);
    dim3 blockK(32, 32, 1);

    int total = 0;
    // Main loop
    while(total <= 10000)
    {
        total++;
        // Calculate distances between all points and all centroids
        hipMemset(d_newCentroid, 0, k * dim * sizeof(float));
        hipMemset(d_count, 0, k * sizeof(int));
        calculateBestDistance2<<<gridN, block>>>(d_tab, d_centroid, d_newCentroid, d_membership, d_count, d_changed, N, k, dim);
        // {
        //     int* debug = new int[k];
        //     hipMemcpy(debug, d_count, k * sizeof(int), hipMemcpyDeviceToHost);
        //     for(int i = 0; i < k; i++)
        //     {
        //         std::cout << i << ") " << debug[i] << std::endl;
        //     }
        // }
        int totalChanged = thrust::reduce(thrust::device, d_changed, d_changed + N, 0);
        //std::cout << "Total changed " << totalChanged << std::endl;
        if(!totalChanged)
            break;
        divNewTab2<<<gridK, blockK>>>(d_centroid, d_newCentroid, d_count, k, dim);
    }
    std::cout << "Total loops for GPU2: " <<  total << std::endl;

    hipMemcpy(h_centroid, d_centroid, dim * k * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_centroid);
    hipFree(d_newCentroid);
    hipFree(d_tab);
    hipFree(d_membership);
    hipFree(d_changed);
    hipFree(d_count);

    return h_centroid;
}